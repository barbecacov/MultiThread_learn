// my_gpu_add.cu
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>

// 核函数
__global__
void vectorAddKernel(float * A_d, float * B_d, float * C_d, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i <= n) C_d[i] = A_d[i] + B_d[i];
}


int main(int argc, char * argv[]){
    int n = atoi(argv[1]);
    std::cout << "执行加法" << n << "次" << std::endl;
    
    // 要申请的内存大小
    size_t size = n * sizeof(float);

    // 先在host上申请内存，生成数据
    // 在CPU上使用 free - malloc 申请内存方式
    float *a = (float*)malloc(size);
    float *b = (float*)malloc(size);
    float *c = (float*)malloc(size);

    // 生成随机数组
    for(int i = 0; i < n; i++){
        float af = rand() / double(RAND_MAX);
        float bf = rand() / double(RAND_MAX);
        a[i] = af;
        b[i] = bf;
    }

    // 定义device指针
    float * da = NULL;
    float * db = NULL;
    float * dc = NULL;

    // 显存申请
    hipMalloc((void **) &da, size);
    hipMalloc((void **) &db, size);
    hipMalloc((void **) &dc, size);

    // 转移拷贝 host --> device
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dc, c, size, hipMemcpyHostToDevice);

    // 调用核函数
    vectorAddKernel<<< n/256+1, 256>>> (da, db, dc, n);
    std::cout<<"nblockPerGrid:"<<n/256 +1 <<"\n";

    // 转移拷贝 device --> host
    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

    // 释放显存
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    
    // std::cout<<"a:"<<*a<<"\n";
    // std::cout<<"b:"<<*b<<"\n";
    // std::cout<<"c:"<<*c<<"\n";
    /*
    $ ./my_gpu_add 1
    a:0.840188
    b:0.394383
    c:1.23457
    通过这个输出可以看出：计算结果是正确的
    */

    // 释放空间
    free(a);
    free(b);
    free(c);
    return 0;
}

